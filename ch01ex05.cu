
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void cudaSayHello()
{
    //get information about blocks and threads
    int tIDx = threadIdx.x;
    int tIDy = threadIdx.y;
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int blockDimX= blockDim.x;
    int blockDimY = blockDim.y;
    int gridDimX = gridDim.x;

    int gtID = (blockY*blockDimY + tIDy)*(gridDimX*blockDimX) + blockX*blockDimX + tIDx;

    printf("Thread - ID<%d,%d> Block<%d,%d>, gtID=%d\n",tIDx,tIDy,blockX,blockY, gtID);

}
int main()
{
    //execute by CPU
    printf("This is from CPU\n");

    //Create 2D grid with 1D block
    dim3 grid(2,4);

    //call the cuda kernel
    //8 blocks are arranged in a 2-by-4 grid
    //1 thread per block
    cudaSayHello<<<grid,1>>>();    

    //Synchronize the cuda device
    hipDeviceSynchronize();

    //end of the program
    return 0;
}