
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void cudaSayHello()
{
    //obtain the global thread ID
    int globalThreadID = blockIdx.x*blockDim.x + threadIdx.x;

    //print the block ID and the thread local ID
    printf("Hello World !, by CUDA - bID = %d,  tID = %d, gtID = %d\n", blockIdx.x, threadIdx.x, globalThreadID);

}
int main()
{
    //execute by CPU
    printf("This is from CPU\n");

    //call the cuda kernel
    cudaSayHello<<<2,3>>>();    //2 block, 3 threads

    //Synchronize the cuda device
    hipDeviceSynchronize();

    //end of the program
    return 0;
}