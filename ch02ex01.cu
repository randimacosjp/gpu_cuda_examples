#include<stdio.h>
#include<hip/hip_runtime.h>

int main()
{
    int deviceCount;

    hipError_t err = hipGetDeviceCount(&deviceCount);

    if(err != hipSuccess)
        printf("CUDA Error : %s", hipGetErrorString(err));
    else
        printf("CUDA enabled device count : %d", deviceCount);

    return 0;
}