#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

int main()
{
    int deviceCount;

    hipError_t err1 = hipGetDeviceCount(&deviceCount);

    if(err1 != hipSuccess)
        printf("CUDA Error : %s", hipGetErrorString(err1));
    else
    {
        printf("\nCUDA enabled device count : %d", deviceCount);
        size_t totalGlobalMemory, freeGlobalMemory;
        hipError_t err2 = hipMemGetInfo(&freeGlobalMemory,&totalGlobalMemory);
        if(err2 != hipSuccess)
            printf("CUDA Error : %s", hipGetErrorString(err2));
        else
            printf("\nFree / Total = %.2f MB/ %.2f MB (%.2f %%)", freeGlobalMemory/pow(2,20),totalGlobalMemory/pow(2,20), float(100*freeGlobalMemory)/totalGlobalMemory);
    }
    printf("\n");
    return 0;
}