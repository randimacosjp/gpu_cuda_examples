#include<stdio.h>
#include<hip/hip_runtime.h>

int main()
{
    int deviceCount;

    hipError_t err = hipGetDeviceCount(&deviceCount);

    if(err != hipSuccess)
        printf("CUDA Error : %s", hipGetErrorString(err));
    else
    {
        //Device property structure
        hipDeviceProp_t devProp;

        //Go through each device
        for(int devNo = 0; devNo<deviceCount; devNo++)
        {
            //extract device properties
            hipError_t err2 = hipGetDeviceProperties(&devProp,devNo);

            if(err2 != hipSuccess)
            {
                return -1;
            }
            else
            {
                printf("\nDevice ID : %d", devNo);
                printf("\nName : %s", devProp.name);
                printf("\nGlobal Mem. size : %d Bytes", devProp.totalGlobalMem);
                printf("\nShared Mem. per block : %d Bytes", devProp.sharedMemPerBlock);
                printf("\nConst. Mem. size : %d Bytes", devProp.totalConstMem);
                printf("\nMax. Threads per block : %d", devProp.maxThreadsPerBlock);
                printf("\nBlock dimension : X %d, Y %d, Z %d", devProp.maxThreadsDim[0],devProp.maxThreadsDim[1],devProp.maxThreadsDim[2]);
                printf("\nGrid dimension : X %d, Y %d, Z %d", devProp.maxGridSize[0],devProp.maxGridSize[1],devProp.maxGridSize[2]);
                printf("\nWarp size %d threads", devProp.warpSize);
                printf("\nMulti Processor Count : %d", devProp.multiProcessorCount);

            }
        }

    }
        
    printf("\n");

    return 0;
}