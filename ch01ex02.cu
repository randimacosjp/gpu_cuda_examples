
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void cudaSayHello()
{
    printf("Hello World !, by CUDA\n");
}
int main()
{
    //execute by CPU
    printf("This is from CPU\n");

    //call the cuda kernel
    cudaSayHello<<<1,5>>>();    //1 block, 5 threads

    //Synchronize the cuda device
    hipDeviceSynchronize();

    //end of the program
    return 0;
}