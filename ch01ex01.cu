
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void cudaSayHello()
{
    printf("Hello World !, by CUDA\n");
}
int main()
{
    //execute by CPU
    printf("This is from CPU\n");

    //call the cuda kernel
    cudaSayHello<<<1,1>>>();    //1 block, 1 threads

    //Synchronize the cuda device
    hipDeviceSynchronize();

    //end of the program
    return 0;
}