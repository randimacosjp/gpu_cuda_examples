
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void cudaSayHello()
{
    //get information about blocks and threads
    int tIDx = threadIdx.x;
    int tIDy = threadIdx.y;
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int blockDimX= blockDim.x;
    int blockDimY = blockDim.y;
    int gridDimX = gridDim.x;

    int gtID = (blockY*blockDimY + tIDy)*(gridDimX*blockDimX) + blockX*blockDimX + tIDx;

    printf("Thread - ID<%d,%d> Block<%d,%d>, gtID=%d\n",tIDx,tIDy,blockX,blockY, gtID);

}
int main()
{
    //execute by CPU
    printf("This is from CPU\n");

    //Create 2D grid with 1D block
    dim3 grid(2,4);	//8 blocks in 2-by-4 grid
    dim3 block(3,2);	//6 threads in 3-by-2 block

    //call the cuda kernel
    cudaSayHello<<<grid,block>>>();    

    //Synchronize the cuda device
    hipDeviceSynchronize();

    //end of the program
    return 0;
}