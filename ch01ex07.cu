
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void cudaSayHello()
{
    //get information about blocks and threads
    int tIDx = threadIdx.x;
    int tIDy = threadIdx.y;
	int tIDz = threadIdx.z;
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
	int blockZ = blockIdx.z;
    int blockDimX= blockDim.x;
    int blockDimY = blockDim.y;
	int blockDimZ = blockDim.z;
	
    int gridDimX = gridDim.x;
	int gridDimY = gridDim.y;
	int gridDimZ = gridDim.z;
	
	printf("Thread (%d,%d,%d) in block (%d,%d,%d)\n", tIDx, tIDy, tIDz, blockX, blockY, blockZ);

}
int main()
{
    //execute by CPU
    printf("This is from CPU\n");

    //Create 3D blocks in 3D grid
    dim3 grid(2,2,2);	//8 blocks in 2x2x2 cubical grid
    dim3 block(2,2,2);	//8 threads in 2x2x2 cubical block

    //call the cuda kernel
    cudaSayHello<<<grid,block>>>();    

    //Synchronize the cuda device
    hipDeviceSynchronize();

    //end of the program
    return 0;
}